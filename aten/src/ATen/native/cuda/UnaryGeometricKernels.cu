#include "hip/hip_runtime.h"
#include <limits>
#include <ATen/native/UnaryOps.h>
#include <ATen/native/cuda/Loops.cuh>
#include <ATen/AccumulateType.h>
#include <ATen/Context.h>
#include <ATen/Dispatch.h>
#include <ATen/native/DispatchStub.h>
#include <ATen/native/TensorIterator.h>

namespace at { namespace native {

void sin_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_FLOATING_AND_COMPLEX_TYPES_AND1(ScalarType::Half, iter.common_dtype(), "sin_cuda", [&]() {
    gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
      return ::sin(a);
    });
  });
}

void cos_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_FLOATING_AND_COMPLEX_TYPES_AND2(ScalarType::Half, ScalarType::BFloat16, iter.common_dtype(), "cos_cuda", [&]() {
    gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
      return ::cos(a);
    });
  });
}

void tan_kernel_cuda(TensorIterator& iter) {
  AT_DISPATCH_FLOATING_AND_COMPLEX_TYPES_AND1(ScalarType::Half, iter.common_dtype(), "tan_cuda", [&]() {
    gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
      return ::tan(a);
    });
  });
}
REGISTER_DISPATCH(sin_stub, &sin_kernel_cuda);
REGISTER_DISPATCH(cos_stub, &cos_kernel_cuda);
REGISTER_DISPATCH(tan_stub, &tan_kernel_cuda);

}} // namespace at::native
